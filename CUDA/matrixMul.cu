#include "hip/hip_runtime.h"
// System includes
#include <assert.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "hip/hip_runtime_api.h"


// mm.cu
#include <hip/hip_runtime.h>


// Basic C++/CUDA explanation
//  https://www.nvidia.com/docs/io/116711/sc11-cuda-c-basics.pdf


// CUDA Kernel, matrix multiply.
//
//      a(M,K) @ b(K,N) = c(M,N)
//
template<int TILE>
__global__
void DeviceMatMul(const float* __restrict__ A,const float* __restrict__ B,float* __restrict__ C,int M,int K,int N)
{
    // Row of C = blockIdx.y*TILE + threadIdx.y
    // Col of C = blockIdx.x*TILE + threadIdx.x
    const int row = blockIdx.y * TILE + threadIdx.y;
    const int col = blockIdx.x * TILE + threadIdx.x;

    // +1 to avoid shared memory bank conflicts on square tiles
    __shared__ float As[TILE][TILE + 1];
    __shared__ float Bs[TILE][TILE + 1];

    float acc = 0.0f;

    // Loop over tiles of K dimension
    for(int tk=0;tk<K;tk+=TILE)
    {
        // Predicated loads (zero-pad out-of-range)
        const int a_col = tk + threadIdx.x;
        const int b_row = tk + threadIdx.y;

        As[threadIdx.y][threadIdx.x] = (row<M && a_col<K) ? A[row*K+a_col] : 0.0f;
        Bs[threadIdx.y][threadIdx.x] = (b_row<K && col<N) ? B[b_row*N+col] : 0.0f;

        __syncthreads();

        // Compute this tile
        #pragma unroll
        for(int k=0;k<TILE;++k)
        {
            acc += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Guarded store
    if(row<M&&col<N)
    {
        C[row*N+col] = acc;
    }
}


// Matrix multiplication - Host code.
//
//      a(M,K) @ b(K,N) = c(M,N)
//
void CudaMatMul(const float* const h_A,const float* const h_B,float* const h_C,const unsigned int M,const unsigned int K,const unsigned int N)
{
    constexpr unsigned int block_size = 32;

    // Allocate host memory for matrices A, B and C.
    float* d_A;
    float* d_B;
    float* d_C;
    unsigned int mem_size_A = M*K*sizeof(float);
    unsigned int mem_size_B = K*N*sizeof(float);
    unsigned int mem_size_C = M*N*sizeof(float);
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_A),mem_size_A));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_B),mem_size_B));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_C),mem_size_C));

	// Create a stream - operations in the stream will synchronize with each other and not block concurrent streams.
    hipStream_t stream;
    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    // Copy host memory to device
    checkCudaErrors(hipMemcpyAsync(d_A,h_A,mem_size_A,hipMemcpyHostToDevice,stream));
    checkCudaErrors(hipMemcpyAsync(d_B,h_B,mem_size_B,hipMemcpyHostToDevice,stream));

    // Setup execution parameters
    dim3 threads(block_size,block_size);
    dim3 grid(((N-1)/block_size)+1,((M-1)/block_size)+1);

    // Call matrix multiply kernal.
    DeviceMatMul<block_size><<<grid,threads,0,stream>>>(d_A,d_B,d_C,M,K,N);
    checkCudaErrors(hipStreamSynchronize(stream));

    // Copy result from device to host
    checkCudaErrors(hipMemcpyAsync(h_C,d_C,mem_size_C,hipMemcpyDeviceToHost,stream));
    checkCudaErrors(hipStreamSynchronize(stream));

    // Free sevice memory
    checkCudaErrors(hipStreamDestroy(stream));
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));
}
